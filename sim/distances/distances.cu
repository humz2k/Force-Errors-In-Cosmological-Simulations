#include <stdio.h>
#include <stdlib.h>
#include "distances.h"
#include "kernels/kernels.h"
#include <hip/hip_fp16.h>

extern "C" {

    void double_precision(double* h_eval_pos, double* h_part_pos, double* h_output, int n_evals, int n_particles){


        double *d_eval_pos, *d_part_pos, *d_output;

        int blockSize = 128;
        int numBlocks = ((n_evals/2) + blockSize - 1) / blockSize;

        hipMalloc(&d_part_pos,n_particles * 3 * sizeof(double));
        hipMalloc(&d_eval_pos,n_evals * 3 * sizeof(double));
        hipMalloc(&d_output,n_evals * n_particles * sizeof(double));

        hipMemcpy(d_part_pos,h_part_pos,n_particles * 3 * sizeof(double),hipMemcpyHostToDevice);
        hipMemcpy(d_eval_pos,h_eval_pos,n_evals * 3 * sizeof(double),hipMemcpyHostToDevice);

        double_distances<<<numBlocks,blockSize>>>(d_eval_pos,d_part_pos,d_output,n_particles,n_evals);

        hipMemcpy(h_output,d_output,n_evals * n_particles * sizeof(double), hipMemcpyDeviceToHost);

        hipFree(d_part_pos);
        hipFree(d_eval_pos);
        hipFree(d_output);

    }

}

extern "C" {

    void single_precision(float* h_eval_pos, float* h_part_pos, float* h_output, int n_evals, int n_particles){


        float *d_eval_pos, *d_part_pos, *d_output;

        int blockSize = 128;
        int numBlocks = (n_evals + blockSize - 1) / blockSize;

        hipMalloc(&d_part_pos,n_particles * 3 * sizeof(float));
        hipMalloc(&d_eval_pos,n_evals * 3 * sizeof(float));
        hipMalloc(&d_output,n_evals * n_particles * sizeof(float));

        hipMemcpy(d_part_pos,h_part_pos,n_particles * 3 * sizeof(float),hipMemcpyHostToDevice);
        hipMemcpy(d_eval_pos,h_eval_pos,n_evals * 3 * sizeof(float),hipMemcpyHostToDevice);

        single_distances<<<numBlocks,blockSize>>>(d_eval_pos,d_part_pos,d_output,n_particles,n_evals);

        hipMemcpy(h_output,d_output,n_evals * n_particles * sizeof(float), hipMemcpyDeviceToHost);

        hipFree(d_part_pos);
        hipFree(d_eval_pos);
        hipFree(d_output);

    }

}

extern "C" {

    void half_precision(half2* h_eval_pos,half2* h_part_pos,half2* h_output,int n_evals,int n_particles){

        half2 *d_eval_pos,*d_part_pos,*d_output;
        //half *d_eval_pos_half,*d_part_pos_half;

        int blockSize = 128;
        int numBlocks = ((n_evals/2) + blockSize - 1) / blockSize;

        hipMalloc(&d_part_pos,(n_particles/2) * 3 * sizeof(half2));
        hipMalloc(&d_eval_pos,(n_evals/2) * 3 * sizeof(half2));

        hipMalloc(&d_output,((n_particles * n_evals)/2) * sizeof(half2));

        hipMemcpy(d_part_pos,h_part_pos,(n_particles/2) * 3 * sizeof(half2),hipMemcpyHostToDevice);
        hipMemcpy(d_eval_pos,h_eval_pos,(n_evals/2) * 3 * sizeof(half2),hipMemcpyHostToDevice);

        half_distances<<<numBlocks,blockSize>>>(d_eval_pos,d_part_pos,d_output,n_particles,n_evals);

        hipMemcpy(h_output,d_output,((n_particles * n_evals)/2) * sizeof(half2), hipMemcpyDeviceToHost);

        hipFree(d_part_pos);
        hipFree(d_eval_pos);
        hipFree(d_output);

    }

}

extern "C" {

    void half_precision_phis(half2* h_eval_pos, half2* h_part_pos, half2* h_masses, half2* G, half2* eps, half2* h_output,int n_evals,int n_particles){

        half2 *d_eval_pos,*d_part_pos,*d_masses,*d_output,*d_G,*d_eps;
        //half *d_eval_pos_half,*d_part_pos_half;

        int blockSize = 128;
        int numBlocks = ((n_evals/2) + blockSize - 1) / blockSize;

        hipMalloc(&d_part_pos,(n_particles/2) * 3 * sizeof(half2));
        hipMalloc(&d_eval_pos,(n_evals/2) * 3 * sizeof(half2));
        hipMalloc(&d_masses,(n_particles/2) * sizeof(half2));

        hipMalloc(&d_G,sizeof(half2));
        hipMalloc(&d_eps,sizeof(half2));

        hipMalloc(&d_output,((n_particles * n_evals)/2) * sizeof(half2));

        hipMemcpy(d_G,G,sizeof(half2),hipMemcpyHostToDevice);
        hipMemcpy(d_eps,eps,sizeof(half2),hipMemcpyHostToDevice);

        hipMemcpy(d_part_pos,h_part_pos,(n_particles/2) * 3 * sizeof(half2),hipMemcpyHostToDevice);
        hipMemcpy(d_eval_pos,h_eval_pos,(n_evals/2) * 3 * sizeof(half2),hipMemcpyHostToDevice);
        hipMemcpy(d_masses,h_masses,(n_particles/2) * sizeof(half2),hipMemcpyHostToDevice);

        //half_phis<<<numBlocks,blockSize>>>(d_eval_pos,d_part_pos,d_masses,G[0],eps[0],d_output,n_particles,n_evals);

        hipMemcpy(h_output,d_output,((n_particles * n_evals)/2) * sizeof(half2), hipMemcpyDeviceToHost);

        hipFree(d_part_pos);
        hipFree(d_eval_pos);
        hipFree(d_output);
        hipFree(d_masses);
        hipFree(d_G);
        hipFree(d_eps);

    }

}