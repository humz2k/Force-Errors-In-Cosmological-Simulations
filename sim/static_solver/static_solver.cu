#include <stdio.h>
#include <stdlib.h>
#include "static_solver.h"
#include "kernels/kernels.h"
#include <hip/hip_fp16.h>

#include <time.h>
#include <sys/time.h>
#define USECPSEC 1000000ULL

using namespace std;

unsigned long long CPUTimer(unsigned long long start=0){

  timeval tv;
  gettimeofday(&tv, 0);
  return ((tv.tv_sec*USECPSEC)+tv.tv_usec)-start;
}

extern "C" {
    void half_precision(half* h_eval_pos, half* h_part_pos, half* h_mass, float* h_output, int n_evals, int n_particles, float G, float eps, unsigned long long* timer){

        hipFree(0);
        hipDeviceSynchronize();

        unsigned long long start,end;

        int blockSize = 128;
        int numBlocks = (n_evals + blockSize - 1) / blockSize;

        half *d_eval_pos, *d_part_pos, *d_mass;
        float *d_output;

        hipMalloc(&d_part_pos,n_particles * 3 * sizeof(half));
        hipMalloc(&d_mass,n_particles * sizeof(half));

        hipMalloc(&d_eval_pos,n_evals * 3 * sizeof(half));
        hipMalloc(&d_output,n_evals * sizeof(float));

        hipMemcpy(d_part_pos,h_part_pos,n_particles * 3 * sizeof(half),hipMemcpyHostToDevice);
        hipMemcpy(d_mass,h_mass,n_particles * sizeof(half),hipMemcpyHostToDevice);

        hipMemcpy(d_eval_pos,h_eval_pos,n_evals * 3 * sizeof(half),hipMemcpyHostToDevice);

        start = CPUTimer();
        half_force_solver<<<numBlocks,blockSize>>>(d_eval_pos,d_part_pos,d_mass,d_output,G,eps,n_particles,n_evals);
        hipDeviceSynchronize();
        end = CPUTimer();

        *timer = end-start;

        hipMemcpy(h_output,d_output,n_evals * sizeof(float),hipMemcpyDeviceToHost);

        hipFree(d_part_pos);
        hipFree(d_mass);
        hipFree(d_output);
        hipFree(d_eval_pos);
    }
}

extern "C" {
    void single_precision(float* h_eval_pos, float* h_part_pos, float* h_mass, float* h_output, int n_evals, int n_particles, float G, float eps, unsigned long long* timer){

        hipFree(0);
        hipDeviceSynchronize();

        unsigned long long start,end;

        int blockSize = 128;
        int numBlocks = (n_evals + blockSize - 1) / blockSize;

        float *d_eval_pos, *d_part_pos, *d_mass, *d_output;

        hipMalloc(&d_part_pos,n_particles * 3 * sizeof(float));
        hipMalloc(&d_mass,n_particles * sizeof(float));

        hipMalloc(&d_eval_pos,n_evals * 3 * sizeof(float));
        hipMalloc(&d_output,n_evals * sizeof(float));

        hipMemcpy(d_part_pos,h_part_pos,n_particles * 3 * sizeof(float),hipMemcpyHostToDevice);
        hipMemcpy(d_mass,h_mass,n_particles * sizeof(float),hipMemcpyHostToDevice);

        hipMemcpy(d_eval_pos,h_eval_pos,n_evals * 3 * sizeof(float),hipMemcpyHostToDevice);

        start = CPUTimer();
        single_force_solver<<<numBlocks,blockSize>>>(d_eval_pos,d_part_pos,d_mass,d_output,G,eps,n_particles,n_evals);
        hipDeviceSynchronize();
        end = CPUTimer();

        *timer = end-start;

        hipMemcpy(h_output,d_output,n_evals * sizeof(float),hipMemcpyDeviceToHost);

        hipFree(d_part_pos);
        hipFree(d_mass);
        hipFree(d_output);
        hipFree(d_eval_pos);
    }
}

extern "C" {
    void double_precision(double* h_eval_pos, double* h_part_pos, double* h_mass, double* h_output, int n_evals, int n_particles, double G, double eps, unsigned long long* timer){

        hipFree(0);
        hipDeviceSynchronize();

        unsigned long long start,end;

        int blockSize = 128;
        int numBlocks = (n_evals + blockSize - 1) / blockSize;

        double *d_eval_pos, *d_part_pos, *d_mass, *d_output;

        hipMalloc(&d_part_pos,n_particles * 3 * sizeof(double));
        hipMalloc(&d_mass,n_particles * sizeof(double));

        hipMalloc(&d_eval_pos,n_evals * 3 * sizeof(double));
        hipMalloc(&d_output,n_evals * sizeof(double));

        hipMemcpy(d_part_pos,h_part_pos,n_particles * 3 * sizeof(double),hipMemcpyHostToDevice);
        hipMemcpy(d_mass,h_mass,n_particles * sizeof(double),hipMemcpyHostToDevice);

        hipMemcpy(d_eval_pos,h_eval_pos,n_evals * 3 * sizeof(double),hipMemcpyHostToDevice);

        start = CPUTimer();
        double_force_solver<<<numBlocks,blockSize>>>(d_eval_pos,d_part_pos,d_mass,d_output,G,eps,n_particles,n_evals);
        hipDeviceSynchronize();
        end = CPUTimer();

        *timer = end-start;

        hipMemcpy(h_output,d_output,n_evals * sizeof(double),hipMemcpyDeviceToHost);

        hipFree(d_part_pos);
        hipFree(d_mass);
        hipFree(d_output);
        hipFree(d_eval_pos);
    }
}

extern "C" {
    void single_precision_shared_mem_cuda(float* h_eval_pos, float* h_part_pos, float* h_mass, float* h_output, int n_evals, int n_particles, float G, float eps, unsigned long long* timer){

        hipFree(0);
        hipDeviceSynchronize();

        unsigned long long start,end;

        int blockSize = 128;
        int numBlocks = (n_evals + blockSize - 1) / blockSize;

        size_t shared_mem_size = blockSize * 4 * sizeof(float);

        float *d_eval_pos, *d_part_pos, *d_mass, *d_output;

        hipMalloc(&d_part_pos,n_particles * 3 * sizeof(float));
        hipMalloc(&d_mass,n_particles * sizeof(float));

        hipMalloc(&d_eval_pos,n_evals * 3 * sizeof(float));
        hipMalloc(&d_output,n_evals * sizeof(float));

        hipMemcpy(d_part_pos,h_part_pos,n_particles * 3 * sizeof(float),hipMemcpyHostToDevice);
        hipMemcpy(d_mass,h_mass,n_particles * sizeof(float),hipMemcpyHostToDevice);

        hipMemcpy(d_eval_pos,h_eval_pos,n_evals * 3 * sizeof(float),hipMemcpyHostToDevice);

        start = CPUTimer();
        single_force_solver_shared_mem_cuda<<<numBlocks,blockSize,shared_mem_size>>>(d_eval_pos,d_part_pos,d_mass,d_output,G,eps,n_particles,n_evals);
        hipDeviceSynchronize();
        end = CPUTimer();

        *timer = end-start;

        hipMemcpy(h_output,d_output,n_evals * sizeof(float),hipMemcpyDeviceToHost);

        hipFree(d_part_pos);
        hipFree(d_mass);
        hipFree(d_output);
        hipFree(d_eval_pos);
    }
}

extern "C" {
    void half_precision_shared_mem_cuda(half* h_eval_pos, half* h_part_pos, half* h_mass, float* h_output, int n_evals, int n_particles, float G, float eps, unsigned long long* timer){

        hipFree(0);
        hipDeviceSynchronize();

        unsigned long long start,end;

        int blockSize = 128;
        int numBlocks = (n_evals + blockSize - 1) / blockSize;

        size_t shared_mem_size = blockSize * 2 * sizeof(half2);

        half *d_eval_pos, *d_part_pos, *d_mass;
        float *d_output;

        hipMalloc(&d_part_pos,n_particles * 3 * sizeof(half));
        hipMalloc(&d_mass,n_particles * sizeof(half));

        hipMalloc(&d_eval_pos,n_evals * 3 * sizeof(half));
        hipMalloc(&d_output,n_evals * sizeof(float));

        hipMemcpy(d_part_pos,h_part_pos,n_particles * 3 * sizeof(half),hipMemcpyHostToDevice);
        hipMemcpy(d_mass,h_mass,n_particles * sizeof(half),hipMemcpyHostToDevice);

        hipMemcpy(d_eval_pos,h_eval_pos,n_evals * 3 * sizeof(half),hipMemcpyHostToDevice);

        start = CPUTimer();
        half_force_solver_shared_mem_cuda<<<numBlocks,blockSize,shared_mem_size>>>(d_eval_pos,d_part_pos,d_mass,d_output,G,eps,n_particles,n_evals);
        hipDeviceSynchronize();
        end = CPUTimer();

        *timer = end-start;

        hipMemcpy(h_output,d_output,n_evals * sizeof(float),hipMemcpyDeviceToHost);

        hipFree(d_part_pos);
        hipFree(d_mass);
        hipFree(d_output);
        hipFree(d_eval_pos);
    }
}